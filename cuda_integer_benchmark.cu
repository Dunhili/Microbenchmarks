#include "hip/hip_runtime.h"
/*
 * Author: Brian Bowden
 * Date: 1/25/14
 *
 * cuda_integer_benchmark.cu
 * 
 * Microbenchmarks designed to test find the throughput of ints, u_ints, floats, and doubles.
 * The kernels for each instruction are designed so that the compiler doesn't optimize the
 * instructions out and each kernel will perform each instruction as we want. The times are
 * normalized using the clock speed of the GPU and the number of instructions to get an 
 * instruction/cycle result. 
 */

#include <stdlib.h>
#include <stdio.h>
#include "repeat.h"

#define REPEAT(iters)	repeat ## iters

void print_results(double average_time, int iterations);
void getThroughput(int benchmark, int iterations);
void gpu_init();
int gcd(int a, int b);

enum Data_Types {
    INT, UINT, FLOAT, DOUBLE
};

enum Instructions {
    Add, Sub, Mul, Div, MAD, VAdd, AND, OR, XOR, SHL, SHR, LRot, RRot
};

// change two lines below if you want to test Integers or Unsigned Integers
typedef int TYPE;
#define DATATYPE (INT)

// constants
const int number_runs = 25;
const int instructions_per_repeat = 4;

// updated in the gpu_init function
float clock_speed;
int number_multi_processors;
int number_blocks;
int number_threads;
int max_threads_per_mp;
int block_size;

// host arrays
TYPE* host_A;
TYPE* host_B;
TYPE* host_C;
TYPE* host_D;

// device arrays
TYPE* device_A;
TYPE* device_B;
TYPE* device_C;
TYPE* device_D;
	
hipEvent_t start, stop;

__global__ void kernelAdd(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat4096(a_val += b_val;
               b_val += c_val;
               c_val += d_val;
               d_val += a_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelSub(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    TYPE e_val = E[i];
    repeat4096(a_val -= b_val;
               b_val -= c_val;
               c_val -= d_val;
               d_val -= e_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelMul(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat4096(a_val *= b_val;
               b_val *= c_val;
               c_val *= d_val;
               d_val *= a_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelDiv(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat256(a_val /= b_val;
              b_val /= c_val;
              c_val /= d_val;
              d_val /= a_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelMAD(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat4096(a_val *= b_val; a_val += b_val;
               b_val *= c_val; b_val += c_val;
               c_val *= d_val; c_val += d_val;
               d_val *= a_val; d_val += a_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelVectorAdd(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat2048(a_val += b_val + c_val;
               b_val += c_val + d_val;
               c_val += d_val + a_val;
               d_val += a_val + b_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelRemainder(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];	
    repeat256(a_val %= b_val;
              b_val %= c_val;
              c_val %= d_val;
              d_val %= a_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelAND(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat4096(a_val = b_val & c_val;
               b_val = c_val & d_val;
               c_val = d_val & a_val;
               d_val = a_val & b_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelOR(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat4096(a_val = b_val | c_val;
               b_val = c_val | d_val;
               c_val = d_val | a_val;
               d_val = a_val | b_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelXOR(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat4096(a_val = b_val ^ c_val;
               b_val = c_val ^ d_val;
               c_val = d_val ^ a_val;
               d_val = a_val ^ b_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelShl(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat4096(a_val <<= b_val;
               b_val <<= c_val;
               c_val <<= d_val;
               d_val <<= a_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelShr(TYPE* A, TYPE* B, TYPE* C, TYPE* D) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat4096(a_val >>= b_val;
               b_val >>= c_val;
               c_val >>= d_val;
               d_val >>= a_val;);
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelLeftRotate(TYPE* A, TYPE* B, TYPE* C, TYPE* D, int shift) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat1024(a_val = (b_val << shift) | (b_val >> (32 - shift)); 
               b_val = (c_val << shift) | (c_val >> (32 - shift)); 
               c_val = (d_val << shift) | (d_val >> (32 - shift)); 
               d_val = (a_val << shift) | (a_val >> (32 - shift)););
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

__global__ void kernelRightRotate(TYPE* A, TYPE* B, TYPE* C, TYPE* D, int shift) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    TYPE a_val = A[i];
    TYPE b_val = B[i];
    TYPE c_val = C[i];
    TYPE d_val = D[i];
    repeat1024(a_val = (b_val >> shift) | (b_val << (32 - shift)); 
               b_val = (c_val >> shift) | (c_val << (32 - shift)); 
               c_val = (d_val >> shift) | (d_val << (32 - shift)); 
               d_val = (a_val >> shift) | (a_val << (32 - shift)););
    A[i] = a_val;
    B[i] = b_val;
    C[i] = c_val;
    D[i] = d_val;
}

/*
 * Prints out the results for the current throughput test. 
 */
void print_results(double average_time, int iterations) {
    int number_instructions = max_threads_per_mp * number_multi_processors * iterations * instructions_per_repeat;
    long number_cycles = (long) ((average_time / 1000) * clock_speed);
    double throughput = ((double) number_instructions) / ((double) number_cycles);
    printf("%0.3g\n", throughput);
}

/*
 * Prints out and calls the appropriate throughput test.
 */
void getThroughput(Instructions instr, int iterations) {
    switch (instr) {
        case Add:  printf("Addition:        "); break;
        case Sub:  printf("Subtraction:     "); break;
        case Mul:  printf("Multiplication:  "); break;
        case Div:  printf("Division:        "); break;
        case MAD:  printf("Multiply-Add:    "); break;
        case VAdd: printf("Vector-Addition: "); break;
        case Rem:  printf("Remainder:       "); break;
        case AND:  printf("AND:             "); break;
        case OR:   printf("OR:              "); break;
        case XOR:  printf("XOR:             "); break;
        case SHL:  printf("Shift-Left:      "); break;
        case SHR:  printf("Shift-Right:     "); break;
        case LRot: printf("Left-Rotate:     "); break;
        case RRot: printf("Right-Rotate:    "); break;
    }

    double average_time = 0.0;
    float time_elapsed;

    //int shift = 8;
    for (int j = 0; j < number_runs; j++) {
        hipEventRecord(start, 0);
        switch (instr) {
            case Add:  kernelAdd<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);                  break;	
            case Sub:  kernelSub<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);                  break;	
            case Mul:  kernelMul<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);                  break;	
            case Div:  kernelDiv<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);                  break;	
            case MAD:  kernelMAD<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);                  break;
            case VAdd: kernelVectorAdd<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);            break;	
            case Rem:  kernelRemainder<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);            break;
            case AND:  kernelAND<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);                  break;
            case OR:   kernelOR<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);                   break;
            case XOR:  kernelXOR<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);                  break;
            case SHL:  kernelShl<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);                  break;
            case SHR:  kernelShr<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D);                  break;
            case LRot: kernelLeftRotate<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D, shift);    break;
            case RRot: kernelRightRotate<<<number_blocks, number_threads>>>(device_A, device_B, device_C, device_D, shift);   break;
        }
	
        hipEventRecord(stop, 0);
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_elapsed, start, stop);
        average_time += time_elapsed;
    }

    print_results(average_time / number_runs, iterations);
}

/**
 * Initializes the global variables by calling the cuda
 */
void gpu_init() {
    hipDeviceProp_t device_prop;
    int device_count;

    hipGetDeviceCount(&device_count);
    if (device_count != 1) {
        printf("Only want to test a single GPU, exiting...\n");
        exit(EXIT_FAILURE);
    }

    if (hipGetDeviceProperties(&device_prop, 0) != hipSuccess) {
        printf("Problem getting properties for device, exiting...\n");
        exit(EXIT_FAILURE);
    } 

    number_threads = device_prop.maxThreadsPerBlock;
    number_multi_processors = device_prop.multiProcessorCount;
    max_threads_per_mp = device_prop.maxThreadsPerMultiProcessor;
    
    block_size = (max_threads_per_mp / gcd(max_threads_per_mp, number_threads));
    number_threads = max_threads_per_mp / block_size;
    number_blocks = number_multi_processors * block_size;
    clock_speed = device_prop.memoryClockRate * 1000;
}

int gcd(int a, int b) {
    if (a == 0)
	    return b;
    return gcd (b % a, a);
}

int main(int argc, char **argv) {
    gpu_init();
    const int N = max_threads_per_mp * number_multi_processors;
    size_t array_size = N * sizeof(TYPE);
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate host arrays
    host_A = (TYPE *) malloc(array_size);
    host_B = (TYPE *) malloc(array_size);
    host_C = (TYPE *) malloc(array_size);
    host_D = (TYPE *) malloc(array_size);

    if (host_A == NULL || host_B == NULL || host_C == NULL || host_D == NULL) {
        printf("Failed allocating array(s), exiting...\n");
        exit(EXIT_FAILURE);
    }

    //Initilize arrays
    for (int i = 0; i < N; i++) {
        host_A[i] = i * 10000;
        host_B[i] = i * 1000;
        host_C[i] = i * 100;
        host_D[i] = i * 10;
    }

    // Allocate device arrays
    hipMalloc((void**) &device_A, array_size);
    hipMalloc((void**) &device_B, array_size);
    hipMalloc((void**) &device_C, array_size);
    hipMalloc((void**) &device_D, array_size);

    // Copy ints from host to device arrays
    hipMemcpy(device_A, host_A, array_size, hipMemcpyHostToDevice);
    hipMemcpy(device_B, host_B, array_size, hipMemcpyHostToDevice);
    hipMemcpy(device_C, host_C, array_size, hipMemcpyHostToDevice);
    hipMemcpy(device_D, host_D, array_size, hipMemcpyHostToDevice);

    switch(DATATYPE) {
        case INT:    printf("Integer\n");          break;
        case UINT:   printf("Unsigned-Integer\n"); break;
        case FLOAT:  printf("Float\n");            break;
        case DOUBLE: printf("Double\n");           break;
    }

    getThroughput(Add,  4096);
    getThroughput(Sub,  4096);
    getThroughput(Mul,  4096);
    getThroughput(Div,   256);
    getThroughput(MAD,  4096);
    getThroughput(VAdd, 2048);
    getThroughput(Rem,   256);
    getThroughput(AND,  4096);
    getThroughput(OR,   4096);
    getThroughput(XOR,  4096);
    getThroughput(SHL,  4096);
    getThroughput(SHR,  4096);
    getThroughput(LRot, 1024);
    getThroughput(RRot, 1024);

    // Free arrays from memory
    free(host_A);
    free(host_B);
    free(host_C);
    free(host_D);

    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);
    hipFree(device_D);

    return EXIT_SUCCESS;
}